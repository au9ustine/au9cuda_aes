#include "hip/hip_runtime.h"
#include "aes.h"

__global__ void aca_add_round_key(uint32_t *state, uint32_t *key)
{
  size_t i = threadIdx.x;
  state[i] ^= key[i];
}
