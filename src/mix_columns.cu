#include "hip/hip_runtime.h"
#include "aes.h"
__global__ void aca_mix_columns(aca_word_t *state)
{
  aca_word_t col  = threadIdx.x;
  aca_word_t base = col << 2;
  aca_word_t t, Tmp, Tm;

  t = state[base];
  Tmp = state[base] ^ state[base + 1] ^ state[base + 2] ^ state[base + 3];
  Tm = state[base] ^ state[base + 1]; Tm = xtime_byte(Tm) & 0xff; state[base] ^= Tm ^ Tmp;
  Tm = state[base + 1] ^ state[base + 2]; Tm = xtime_byte(Tm) & 0xff; state[base + 1] ^= Tm ^ Tmp;
  Tm = state[base + 2] ^ state[base + 3]; Tm = xtime_byte(Tm) & 0xff; state[base + 2] ^= Tm ^ Tmp;
  Tm = state[base + 3] ^ t;      Tm = xtime_byte(Tm) & 0xff; state[base + 3] ^= Tm ^ Tmp;
}

__global__ void aca_inv_mix_columns(aca_word_t *state)
{
  aca_word_t col = threadIdx.x;
  aca_word_t base = col << 2;
  aca_word_t t, Tmp;
  aca_word_t u, v, w;

  Tmp = state[base] ^ state[base + 1] ^ state[base + 2] ^ state[base + 3];
  u = xtime_byte(Tmp) & 0xff;
  v = xtime_byte(u) & 0xff;
  w = xtime_byte(v) & 0xff;

  t = w ^ Tmp;
  t ^= (xtime_byte((xtime_byte(state[base]) & 0xff)) & 0xff) ^ state[base];
  t ^= (xtime_byte(state[base + 1]) & 0xff);
  t ^= (xtime_byte((xtime_byte(state[base + 2]) & 0xff)) & 0xff);
  state[base] = t;

  t = w ^ Tmp;
  t ^= (xtime_byte((xtime_byte(state[base+1]) & 0xff)) & 0xff) ^ state[base+1];
  t ^= (xtime_byte(state[base+2]) & 0xff);
  t ^= (xtime_byte((xtime_byte(state[base+3]) & 0xff)) & 0xff);
  state[base+1] = t;

  t = w ^ Tmp;
  t ^= (xtime_byte((xtime_byte(state[base+2]) & 0xff)) & 0xff) ^ state[base+2];
  t ^= (xtime_byte(state[base + 3]) & 0xff);
  t ^= (xtime_byte((xtime_byte(state[base]) & 0xff)) & 0xff);
  state[base+2] = t;

  t = w ^ Tmp;
  t ^= (xtime_byte((xtime_byte(state[base+3]) & 0xff)) & 0xff) ^ state[base+3];
  t ^= (xtime_byte(state[base]) & 0xff);
  t ^= (xtime_byte((xtime_byte(state[base+1]) & 0xff)) & 0xff);
  state[base+3] = t;
}
