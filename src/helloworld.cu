
#include <hip/hip_runtime.h>
#include <iostream>

int main(int argc, char *argv[])
{
  // Build a device property
  struct hipDeviceProp_t prop;

  // Get the device property info
  hipGetDeviceProperties(&prop, 0);

  // write to output
  std::cout << prop.name << " says: ";
  std::cout << "Hello, World!" << std::endl;

  return 0;
}
